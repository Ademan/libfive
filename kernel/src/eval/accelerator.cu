#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2016 Matthew Keeter  <matt.j.keeter@gmail.com>
 *
 *  This file is part of the Ao library.
 *
 *  This library is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU Lesser General Public
 *  License as published by the Free Software Foundation; either
 *  version 2.1 of the License, or (at your option) any later version.
 *
 *  Ao is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with Ao.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <cstdio>

#include "ao/kernel/eval/accelerator.hpp"

#include "ao/kernel/eval/evaluator.hpp"
#include "ao/kernel/eval/clause.hpp"

// Helpful macros that are repeated in every kernel
#define GET_INDEX int i = blockIdx.x
#define KERNEL(name) __global__ void name(float* a, float* b, float* out)

////////////////////////////////////////////////////////////////////////////////
// Floating-point kernels
KERNEL(add_f)
{
    GET_INDEX;
    out[i] = a[i] + b[i];
}

KERNEL(mul_f)
{
    GET_INDEX;
    out[i] = a[i] * b[i];
}

KERNEL(min_f)
{
    GET_INDEX;
    out[i] = min(a[i], b[i]);
}

KERNEL(max_f)
{
    GET_INDEX;
    out[i] = max(a[i], b[i]);
}

KERNEL(sub_f)
{
    GET_INDEX;
    out[i] = a[i] - b[i];
}

KERNEL(div_f)
{
    GET_INDEX;
    out[i] = a[i] / b[i];
}

KERNEL(atan2_f)
{
    GET_INDEX;
    out[i] = atan2(a[i], b[i]);
}

KERNEL(mod_f)
{
    GET_INDEX;
    out[i] = fmod(a[i], b[i]);
    while (out[i] < 0)
    {
        out[i] += b[i];
    }
}

KERNEL(nanfill_f)
{
    GET_INDEX;
    out[i] = isnan(a[i]) ? b[i] : a[i];
}

KERNEL(square_f)
{
    GET_INDEX;
    out[i] = a[i] * a[i];
}

KERNEL(sqrt_f)
{
    GET_INDEX;
    out[i] = sqrt(a[i]);
}

KERNEL(neg_f)
{
    GET_INDEX;
    out[i] = -a[i];
}

KERNEL(abs_f)
{
    GET_INDEX;
    out[i] = fabs(a[i]);
}

KERNEL(sin_f)
{
    GET_INDEX;
    out[i] = sin(a[i]);
}

KERNEL(cos_f)
{
    GET_INDEX;
    out[i] = cos(a[i]);
}

KERNEL(tan_f)
{
    GET_INDEX;
    out[i] = tan(a[i]);
}

KERNEL(asin_f)
{
    GET_INDEX;
    out[i] = asin(a[i]);
}

KERNEL(acos_f)
{
    GET_INDEX;
    out[i] = acos(a[i]);
}

KERNEL(atan_f)
{
    GET_INDEX;
    out[i] = atan(a[i]);
}

KERNEL(exp_f)
{
    GET_INDEX;
    out[i] = exp(a[i]);
}

KERNEL(a_f)
{
    GET_INDEX;
    out[i] = a[i];
}

KERNEL(b_f)
{
    GET_INDEX;
    out[i] = b[i];
}

////////////////////////////////////////////////////////////////////////////////

// Pointers are into device memory, not host memory!
static void clause(Opcode op, float* a, float* b, float* out, size_t count)
{
    switch (op) {
        case OP_ADD:    add_f<<<count,1>>>(a, b, out); break;
        case OP_MUL:    mul_f<<<count,1>>>(a, b, out); break;
        case OP_MIN:    min_f<<<count,1>>>(a, b, out); break;
        case OP_MAX:    max_f<<<count,1>>>(a, b, out); break;
        case OP_SUB:    sub_f<<<count,1>>>(a, b, out); break;
        case OP_DIV:    div_f<<<count,1>>>(a, b, out); break;
        case OP_ATAN2:  atan2_f<<<count,1>>>(a, b, out); break;
        case OP_MOD:    mod_f<<<count,1>>>(a, b, out); break;
        case OP_NANFILL:    nanfill_f<<<count,1>>>(a, b, out); break;

        case OP_SQUARE: square_f<<<count,1>>>(a, b, out); break;
        case OP_SQRT: sqrt_f<<<count,1>>>(a, b, out); break;
        case OP_NEG: neg_f<<<count,1>>>(a, b, out); break;
        case OP_ABS: abs_f<<<count,1>>>(a, b, out); break;
        case OP_SIN: sin_f<<<count,1>>>(a, b, out); break;
        case OP_COS: cos_f<<<count,1>>>(a, b, out); break;
        case OP_TAN: tan_f<<<count,1>>>(a, b, out); break;
        case OP_ASIN: asin_f<<<count,1>>>(a, b, out); break;
        case OP_ACOS: acos_f<<<count,1>>>(a, b, out); break;
        case OP_ATAN: atan_f<<<count,1>>>(a, b, out); break;
        case OP_EXP: exp_f<<<count,1>>>(a, b, out); break;

        case OP_A: a_f<<<count,1>>>(a, b, out); break;
        case OP_B: b_f<<<count,1>>>(a, b, out); break;

        case INVALID:
        case OP_CONST:
        case OP_X:
        case OP_Y:
        case OP_Z:
        case AFFINE:
        case LAST_OP: assert(false);
    }
}

////////////////////////////////////////////////////////////////////////////////

Accelerator::Accelerator(Evaluator* e)
    : evaluator(e)
{
    for (auto c : {e->X, e->Y, e->Z})
    {
        allocate(c);
    }
    for (auto c : e->matrix)
    {
        allocate(c);
    }
    for (auto c : e->constants)
    {
        allocate(c);
    }
    for (auto row : e->rows)
    {
        for (auto c : row)
        {
            allocate(c);
        }
    }
}

Accelerator::~Accelerator()
{
    for (auto m : mem)
    {
        hipFree(m.second);
    }
}

////////////////////////////////////////////////////////////////////////////////

void Accelerator::allocate(Clause* c)
{
    const size_t bytes = N * sizeof(float);
    hipMalloc((void**)&mem[c], bytes);

    // If this is a constant operation, fill with the constant value
    if (c->op == OP_CONST)
    {
        std::fill(buf.begin(), buf.end(), c->value);
        hipMemcpy(mem[c], &buf[0], bytes, hipMemcpyHostToDevice);
    }
}

float* Accelerator::values(size_t count)
{
    for (const auto& row : evaluator->rows)
    {
        for (size_t i=0; i < row.active; ++i)
        {
            auto op = row[i]->op;

            // Modify the opcode if parts of the tree are disabled
            if (row[i]->a && row[i]->a->flags & CLAUSE_FLAG_DISABLED)
            {
                op = OP_B;
            }
            if (row[i]->b && row[i]->b->flags & CLAUSE_FLAG_DISABLED)
            {
                op = OP_A;
            }

            clause(op, mem[row[i]->a], mem[row[i]->b], mem[row[i]], count);
        }
    }

    return mem[evaluator->root];
}

void Accelerator::toDevice()
{
    const size_t bytes = N * sizeof(float);
    hipMemcpy(mem[evaluator->X], &X[0], bytes, hipMemcpyHostToDevice);
    hipMemcpy(mem[evaluator->Y], &Y[0], bytes, hipMemcpyHostToDevice);
    hipMemcpy(mem[evaluator->Z], &Z[0], bytes, hipMemcpyHostToDevice);
}

float* Accelerator::fromDevice(float* ptr_d)
{
    const size_t bytes = N * sizeof(float);
    hipMemcpy(&buf[0], ptr_d, bytes, hipMemcpyDeviceToHost);
    return &buf[0];
}
