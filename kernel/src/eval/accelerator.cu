#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2016 Matthew Keeter  <matt.j.keeter@gmail.com>
 *
 *  This file is part of the Ao library.
 *
 *  This library is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU Lesser General Public
 *  License as published by the Free Software Foundation; either
 *  version 2.1 of the License, or (at your option) any later version.
 *
 *  Ao is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with Ao.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <cstdio>
#include <numeric>

#include "ao/kernel/eval/accelerator.hpp"

#include "ao/kernel/eval/evaluator.hpp"
#include "ao/kernel/eval/clause.hpp"
#include "ao/kernel/render/subregion.hpp"

// Helpful macros that are repeated in every kernel
#define GET_INDEX int i = threadIdx.x + blockIdx.x * blockDim.x;\
                  if (i >= Accelerator::N)  return
#define KERNEL(name) __global__ void name(float* __restrict__ a, float* __restrict__ b, float* __restrict__ out)

#define RENDER_PARAMS count/1024,1024
////////////////////////////////////////////////////////////////////////////////
// Floating-point kernels
KERNEL(add_f)
{
    GET_INDEX;
    out[i] = a[i] + b[i];
}

KERNEL(mul_f)
{
    GET_INDEX;
    out[i] = a[i] * b[i];
}

KERNEL(min_f)
{
    GET_INDEX;
    out[i] = min(a[i], b[i]);
}

KERNEL(max_f)
{
    GET_INDEX;
    out[i] = max(a[i], b[i]);
}

KERNEL(sub_f)
{
    GET_INDEX;
    out[i] = a[i] - b[i];
}

KERNEL(div_f)
{
    GET_INDEX;
    out[i] = a[i] / b[i];
}

KERNEL(atan2_f)
{
    GET_INDEX;
    out[i] = atan2(a[i], b[i]);
}

KERNEL(mod_f)
{
    GET_INDEX;
    out[i] = fmod(a[i], b[i]);
    while (out[i] < 0)
    {
        out[i] += b[i];
    }
}

KERNEL(nanfill_f)
{
    GET_INDEX;
    out[i] = isnan(a[i]) ? b[i] : a[i];
}

KERNEL(square_f)
{
    GET_INDEX;
    out[i] = a[i] * a[i];
}

KERNEL(sqrt_f)
{
    GET_INDEX;
    out[i] = sqrt(a[i]);
}

KERNEL(neg_f)
{
    GET_INDEX;
    out[i] = -a[i];
}

KERNEL(abs_f)
{
    GET_INDEX;
    out[i] = fabs(a[i]);
}

KERNEL(sin_f)
{
    GET_INDEX;
    out[i] = sin(a[i]);
}

KERNEL(cos_f)
{
    GET_INDEX;
    out[i] = cos(a[i]);
}

KERNEL(tan_f)
{
    GET_INDEX;
    out[i] = tan(a[i]);
}

KERNEL(asin_f)
{
    GET_INDEX;
    out[i] = asin(a[i]);
}

KERNEL(acos_f)
{
    GET_INDEX;
    out[i] = acos(a[i]);
}

KERNEL(atan_f)
{
    GET_INDEX;
    out[i] = atan(a[i]);
}

KERNEL(exp_f)
{
    GET_INDEX;
    out[i] = exp(a[i]);
}

KERNEL(a_f)
{
    GET_INDEX;
    out[i] = a[i];
}

KERNEL(b_f)
{
    GET_INDEX;
    out[i] = b[i];
}

__global__ void flatten_region(float* x, float xmin, float xmax, int ni,
                               float* y, float ymin, float ymax, int nj,
                               float* z, float zmin, float zmax, int nk)
{
    int index = blockIdx.x;
    int i = index / (nj * nk);
    int j = (index / nk) % nj;
    int k = index % nk;

    if (i <= ni)
    {
        float frac = (i + 0.5f) / ni;
        x[index] = xmin * (1.0f - frac) + xmax * frac;
    }
    if (j <= nj)
    {
        float frac = (j + 0.5f) / nj;
        y[index] = ymin * (1.0f - frac) + ymax * frac;
    }
    if (k <= nk)
    {
        float frac = (k + 0.5f) / nk;
        z[index] = zmin * (1.0f - frac) + zmax * frac;
    }
}

////////////////////////////////////////////////////////////////////////////////

// Pointers are into device memory, not host memory!
static void clause(Opcode op, float* a, float* b, float* out, size_t count)
{
    switch (op) {
        case OP_ADD:    add_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_MUL:    mul_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_MIN:    min_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_MAX:    max_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_SUB:    sub_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_DIV:    div_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_ATAN2:  atan2_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_MOD:    mod_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_NANFILL:    nanfill_f<<<RENDER_PARAMS>>>(a, b, out); break;

        case OP_SQUARE: square_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_SQRT: sqrt_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_NEG: neg_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_ABS: abs_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_SIN: sin_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_COS: cos_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_TAN: tan_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_ASIN: asin_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_ACOS: acos_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_ATAN: atan_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_EXP: exp_f<<<RENDER_PARAMS>>>(a, b, out); break;

        case OP_A: a_f<<<RENDER_PARAMS>>>(a, b, out); break;
        case OP_B: b_f<<<RENDER_PARAMS>>>(a, b, out); break;

        case INVALID:
        case OP_CONST:
        case OP_X:
        case OP_Y:
        case OP_Z:
        case AFFINE:
        case LAST_OP: assert(false);
    }
}

////////////////////////////////////////////////////////////////////////////////

Accelerator::Accelerator(Evaluator* e)
    : evaluator(e)
{
    // Count up the number of clauses in the evaluator
    size_t count =  std::accumulate(e->rows.begin(), e->rows.end(),
            3                           // X, Y, Z
            + e->constants.size(),      // Constants
            [](size_t i, const Row& r){ return i + r.size(); });

    auto out = hipMalloc((void**)&data, N * count * sizeof(float));

    // If this is a constant operation, fill with the constant value
    for (auto c : e->constants)
    {
        std::fill(buf.begin(), buf.end(), c->value);
        hipMemcpy(devPtr(c), &buf[0], N * sizeof(float),
                   hipMemcpyHostToDevice);
    }
}

Accelerator::~Accelerator()
{
    hipFree(data);
}

////////////////////////////////////////////////////////////////////////////////

float* Accelerator::devPtr(const Clause* c)
{
    // Clauses are allocated in the data array of the evaluator
    // (using placement new); this function finds the clause's
    // location in that array then maps to the device data array
    return &data[(c - evaluator->data) * N];
}

////////////////////////////////////////////////////////////////////////////////

float* Accelerator::values(size_t count)
{
    for (const auto& row : evaluator->rows)
    {
        for (size_t i=0; i < row.active; ++i)
        {
            auto op = row[i]->op;

            // Modify the opcode if parts of the tree are disabled
            if (row[i]->a && row[i]->a->flags & CLAUSE_FLAG_DISABLED)
            {
                op = OP_B;
            }
            if (row[i]->b && row[i]->b->flags & CLAUSE_FLAG_DISABLED)
            {
                op = OP_A;
            }
            clause(op, devPtr(row[i]->a), devPtr(row[i]->b), devPtr(row[i]), count);
        }
    }

    return devPtr(evaluator->root);
}

void Accelerator::toDevice()
{
    const size_t bytes = N * sizeof(float);
    hipMemcpy(devPtr(evaluator->X), &X[0], bytes, hipMemcpyHostToDevice);
    hipMemcpy(devPtr(evaluator->Y), &Y[0], bytes, hipMemcpyHostToDevice);
    hipMemcpy(devPtr(evaluator->Z), &Z[0], bytes, hipMemcpyHostToDevice);
}

float* Accelerator::fromDevice(float* ptr_d)
{
    const size_t bytes = N * sizeof(float);
    hipMemcpy(&buf[0], ptr_d, bytes, hipMemcpyDeviceToHost);
    return &buf[0];
}

void Accelerator::warmup()
{
    float* a = new float[N];
    float* b = new float[N];
    float* out = new float[N];

    float* a_d = nullptr;
    float* b_d = nullptr;
    float* out_d = nullptr;

    auto bytes = N * sizeof(float);
    hipMalloc((void**)&a_d, bytes);
    hipMalloc((void**)&b_d, bytes);
    hipMalloc((void**)&out_d, bytes);

    hipMemcpy(&a_d, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(&b_d, b, bytes, hipMemcpyHostToDevice);

    add_f<<<N, 1>>>(a_d, b_d, out_d);
    hipMemcpy(out, out_d, bytes, hipMemcpyDeviceToHost);

    delete [] a;
    delete [] b;
    delete [] out;

    hipFree(a_d);
    hipFree(b_d);
    hipFree(out_d);
}

void Accelerator::setRegion(const Subregion& r)
{
    flatten_region<<<r.voxels(), 1>>>(
        devPtr(evaluator->X), r.X.lower(), r.X.upper(), r.X.size,
        devPtr(evaluator->Y), r.Y.lower(), r.Y.upper(), r.Y.size,
        devPtr(evaluator->Z), r.Z.lower(), r.Z.upper(), r.Z.size);
}
