#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2016 Matthew Keeter  <matt.j.keeter@gmail.com>
 *
 *  This file is part of the Ao library.
 *
 *  This library is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU Lesser General Public
 *  License as published by the Free Software Foundation; either
 *  version 2.1 of the License, or (at your option) any later version.
 *
 *  Ao is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with Ao.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <iostream>

#include "ao/kernel/cuda/tape.hpp"

#include "ao/kernel/eval/evaluator.hpp"
#include "ao/kernel/eval/clause.hpp"
#include "ao/kernel/render/region.hpp"

#define ARG_A_CONST 0x100
#define ARG_A_MEM   0x200

#define ARG_B_CONST 0x1000
#define ARG_B_MEM   0x2000

/*  This is the tape that the evaluator uses  */
__constant__ int tape_d[TapeAccelerator::NUM_CLAUSES * 2];
__constant__ float constants_d[TapeAccelerator::NUM_CONSTANTS];

__constant__ uint32_t clause_count_d;
__constant__ uint32_t tree_root_d;

/*
 *  Evaluate a single point (see eval below for details)
 */
__device__ float eval_single(float x, float y, float z)
{
    // This is our local slice of memory used to store clause results
    float local[TapeAccelerator::NUM_CLAUSES];

    local[0] = x;
    local[1] = y;
    local[2] = z;

    // Make sure that the tape is ready for use
    __syncthreads();

    // First three clauses are dummies for X, Y, Z coordinates
    for (int clause_index=3; clause_index < clause_count_d; clause_index++)
    {
        // Grab the next opcode from the tape
        uint32_t opcode = tape_d[2 * clause_index];
        uint32_t   addr = tape_d[2 * clause_index + 1];

        uint16_t a_addr = addr & 0xFFFF;
        uint16_t b_addr = (addr >> 16) & 0xFFFF;

        // These are the values that we'll do math on
        float a, b;

        // Read arguments if present, selecting either from an immediate
        // argument (i.e. an inline float) or an address in the local mem
        if (opcode & ARG_A_CONST)
        {
            a = constants_d[a_addr];
        }
        else if (opcode & ARG_A_MEM)
        {
            a = local[a_addr];
        }

        if (opcode & ARG_B_CONST)
        {
            b = constants_d[b_addr];
        }
        else if (opcode & ARG_B_MEM)
        {
            b = local[b_addr];
        }

        switch (opcode & 0xFF)
        {
            case OP_ADD:    local[clause_index] = a + b; break;
            case OP_MUL:    local[clause_index] = a * b; break;

            case OP_MIN:    local[clause_index] = fmin(a, b); break;
            case OP_MAX:    local[clause_index] = fmax(a, b); break;
            case OP_SUB:    local[clause_index] = a - b; break;
            case OP_DIV:    local[clause_index] = a / b; break;
            case OP_ATAN2:  local[clause_index] = atan2(a, b); break;
            case OP_MOD:    local[clause_index] = fmod(a, b);
                            while (local[clause_index] < 0)
                            {
                                local[clause_index] += b;
                            }
                            break;
            case OP_NANFILL:    local[clause_index] = isnan(a) ? b : a; break;

            case OP_SQUARE: local[clause_index] = a * a; break;
            case OP_SQRT:   local[clause_index] = sqrt(a); break;
            case OP_NEG:    local[clause_index] = -a; break;
            case OP_ABS:    local[clause_index] = fabs(a); break;
            case OP_SIN:    local[clause_index] = sin(a); break;
            case OP_COS:    local[clause_index] = cos(a); break;
            case OP_TAN:    local[clause_index] = tan(a); break;
            case OP_ASIN:   local[clause_index] = asin(a); break;
            case OP_ACOS:   local[clause_index] = acos(a); break;
            case OP_ATAN:   local[clause_index] = atan(a); break;
            case OP_EXP:    local[clause_index] = exp(a); break;
        }
    }

    return local[tree_root_d];
}

/*
 *  eval is a kernel that executes an instruction tape in parallel.
 *  We use tape_d as a global constant tape to read
 *      tape_d is a list of opcodes enums or'd with supporting data
 *          The lowest byte of each uint32 is the opcode itself
 *          The third and fourth nibbles are both
 *                  0x0 if this argument isn't relevant
 *                  0x1 if an immediate argument follows
 *                  0x2 if a memory address is given
 *              for the a and b arguments
 *  The arguments are as follows:
 *      X, Y, Z are pointers into device memory storing coordinates
 *      out is a pointer into device memory for the output
 *      clauses is the number of clauses to be evaluated
 *      root is the clause number to be copied to output
 */
__global__ void eval(float const* X, float const* Y, float const* Z,
                     float* out)
{
    // Index of this piece of work in the global space
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Evaluate the expression on the target coordinates
    out[index] = eval_single(X[index], Y[index], Z[index]);
}

__global__ void eval_region(float xmin, float xmax, int imin, int ni,
                            float ymin, float ymax, int jmin, int nj,
                            float zmin, float zmax, int kmin, int nk,
                            uint32_t* image, uint32_t stride)
{
    // Index of this piece of work in the global space
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    int i = index / (nj * nk);
    int j = (index / nk) % nj;
    int k = index % nk;

    // Abort if we're too beyond the bounds of reason
    if (i > ni)
    {
        return;
    }

    // Find our global coordinates
    float x_frac = (i + 0.5f) / ni;
    float x = xmin * (1.0f - x_frac) + xmax * x_frac;

    float y_frac = (j + 0.5f) / nj;
    float y = ymin * (1.0f - y_frac) + ymax * y_frac;

    float z_frac = (k + 0.5f) / nk;
    float z = zmin * (1.0f - z_frac) + zmax * z_frac;

    // Evaluate the expression on the target coordinates
    float out = eval_single(x, y, z);

    // If this reading is less than zero, update the heightmap
    if (out < 0)
    {
        atomicMax(&image[imin + i + (jmin + j) * stride], k);
    }
}

////////////////////////////////////////////////////////////////////////////////

TapeAccelerator::TapeAccelerator(Evaluator* e)
    : Accelerator(e)
{
    // Allocate space for the input and output arrays
    hipMalloc((void**)&X_d, N * sizeof(float));
    hipMalloc((void**)&Y_d, N * sizeof(float));
    hipMalloc((void**)&Z_d, N * sizeof(float));
    hipMalloc((void**)&out_d, N * sizeof(float));

    reloadTape();
}

void TapeAccelerator::reloadTape()
{
    // Construct the tape!
    std::vector<uint32_t> tape = {OP_X, 0, OP_Y, 0, OP_Z, 0};
    std::vector<float> constants;

    uint32_t clause_count = 3;

    // clause_addr stores addresses of normal clauses
    std::unordered_map<Clause*, uint32_t> clause_addr =
        {{evaluator->X, 0}, {evaluator->Y, 1}, {evaluator->Z, 2}};

    // const_addr maps from constant values to addresses
    std::unordered_map<float, uint32_t> const_addr;

    for (const auto& r : evaluator->rows)
    {
        for (size_t i=0; i < r.active; ++i)
        {
            Clause* c = r[i];

            uint32_t op = c->op;
            uint32_t addr = 0;

            clause_addr[c] = clause_count++;

            if (c->a)
            {
                if (c->a->op == OP_CONST)
                {
                    if (const_addr.count(c->a->value) == 0)
                    {
                        const_addr[c->a->value] = constants.size();
                        constants.push_back(c->a->value);
                    }
                    op |= ARG_A_CONST;
                    addr |= const_addr[c->a->value];
                }
                else
                {
                    op |= ARG_A_MEM;
                    addr |= clause_addr[c->a];
                }
            }

            if (c->b)
            {
                if (c->b->op == OP_CONST)
                {
                    if (const_addr.count(c->b->value) == 0)
                    {
                        const_addr[c->b->value] = constants.size();
                        constants.push_back(c->b->value);
                    }
                    op |= ARG_B_CONST;
                    addr |= (const_addr[c->b->value] << 16);
                }
                else
                {
                    op |= ARG_B_MEM;
                    addr |= (clause_addr[c->b] << 16);
                }
            }

            tape.push_back(op);
            tape.push_back(addr);
        }
    }

    assert(clause_addr.count(evaluator->root));

    // Make sure that we didn't run out space in our global arrays
    assert(clause_count <= NUM_CLAUSES);
    assert(constants.size() <= NUM_CONSTANTS);

    // Copy the tape over to the GPU
    hipMemcpyToSymbol(HIP_SYMBOL(tape_d), &tape[0], tape.size() * sizeof(uint32_t));

    // Copy the constant array to the GPU
    hipMemcpyToSymbol(HIP_SYMBOL(constants_d), &constants[0],
                       constants.size() * sizeof(float));

    // Deploy clause count and root address to GPU memory
    uint32_t root = clause_addr[evaluator->root];
    hipMemcpyToSymbol(HIP_SYMBOL(clause_count_d), &clause_count, sizeof(clause_count));
    hipMemcpyToSymbol(HIP_SYMBOL(tree_root_d), &root, sizeof(root));
}

TapeAccelerator::~TapeAccelerator()
{
    for (auto& ptr : {X_d, Y_d, Z_d, out_d})
    {
        hipFree(ptr);
    }

    if (image_d)
    {
        hipFree(image_d);
    }
}

void TapeAccelerator::allocateImage(const Region& r)
{
    image_dims = {r.X.values.size(), r.Y.values.size(), r.Z.values.size()};
    image_min = {r.X.bounds.lower(), r.Y.bounds.lower(), r.Z.bounds.lower()};
    image_max = {r.X.bounds.upper(), r.Y.bounds.upper(), r.Z.bounds.upper()};

    size_t bytes = image_dims.x * image_dims.y * sizeof(uint32_t);
    hipMalloc(&image_d, bytes);
    hipMemset(image_d, 0,  bytes);
}

void TapeAccelerator::render(const Subregion& r)
{
    assert(r.voxels() <= N);

    int blocks = (r.voxels() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    eval_region<<<blocks, THREADS_PER_BLOCK>>>(
        r.X.lower(), r.X.upper(), r.X.min, r.X.size,
        r.Y.lower(), r.Y.upper(), r.Y.min, r.Y.size,
        r.Z.lower(), r.Z.upper(), r.Z.min, r.Z.size,
        image_d, image_dims.x);
}

void TapeAccelerator::getImage() const
{
    auto out = new uint32_t[image_dims.x * image_dims.y];

    hipMemcpy(out, image_d,
               image_dims.x * image_dims.y * sizeof(uint32_t),
               hipMemcpyDeviceToHost);

    int k=0;
    for (int i=0; i < image_dims.y; ++i)
    {
        for (int j=0; j < image_dims.x; ++j)
            std::cout << out[k++] << ' ';
        std::cout << '\n';
    }
}

float* TapeAccelerator::values(size_t count)
{
    int blocks = (count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    eval<<<blocks, THREADS_PER_BLOCK>>>(X_d, Y_d, Z_d, out_d);

    return out_d;
}
