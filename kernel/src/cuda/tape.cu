#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2016 Matthew Keeter  <matt.j.keeter@gmail.com>
 *
 *  This file is part of the Ao library.
 *
 *  This library is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU Lesser General Public
 *  License as published by the Free Software Foundation; either
 *  version 2.1 of the License, or (at your option) any later version.
 *
 *  Ao is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with Ao.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "ao/kernel/cuda/tape.hpp"

#include "ao/kernel/eval/evaluator.hpp"
#include "ao/kernel/eval/clause.hpp"

#define ARG_A_IMM 0x100
#define ARG_A_MEM 0x200

#define ARG_B_IMM 0x1000
#define ARG_B_MEM 0x2000
/*
 *  eval is a kernel that executes an instruction tape in parallel.
 *  The arguments are as follows:
 *      tape is a list of opcodes enums or'd with supporting data
 *          The lowest byte of each uint32 is the opcode itself
 *          The third and fourth nibbles are both
 *                  0x0 if this argument isn't relevant
 *                  0x1 if an immediate argument follows
 *                  0x2 if a memory address is given
 *              for the a and b arguments
 *      X, Y, Z are pointers into device memory storing coordinates
 *      out is a pointer into device memory for the output
 *      mem is a pointer into device memory for scratch results
 *          It should be clauses * blocks * threads * sizeof(float) in size
 *      clauses is the number of clauses to be evaluated
 */
__global__ void eval(uint32_t const* tape,
                     float const* X, float const* Y, float const* Z,
                     float* out, float* mem, int clauses, uint32_t root)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // This is our local slice of the big memory buffer
    float* local = &mem[index * clauses];

    // Load coordinates into the buffer
    local[0] = X[index];
    local[1] = Y[index];
    local[2] = Z[index];

    // First three opcodes are dummies for X, Y, Z coordinates
    int i=3;
    int j=3;
    while(tape[i])
    {
        // Grab the next opcode from the tape
        uint32_t opcode = tape[i++];

        // These are the values that we'll do math on
        float a, b;

        // Read arguments if present, selecting either from an immediate
        // argument (i.e. an inline float) or an address in the local mem
        if (opcode & ARG_A_IMM)
        {
            a = ((float*)tape)[i++];
        }
        else if (opcode & ARG_A_MEM)
        {
            a = local[tape[i++]];
        }

        if (opcode & ARG_B_IMM)
        {
            b = ((float*)tape)[i++];
        }
        else if (opcode & ARG_B_MEM)
        {
            b = local[tape[i++]];
        }

        switch (opcode & 0xFF)
        {
            case OP_ADD:    local[j] = a + b; break;
            case OP_MUL:    local[j] = a * b; break;

            case OP_MIN:    local[j] = fmin(a, b); break;
            case OP_MAX:    local[j] = fmax(a, b); break;
            case OP_SUB:    local[j] = a - b; break;
            case OP_DIV:    local[j] = a / b; break;
            case OP_ATAN2:  local[j] = atan2(a, b); break;
            case OP_MOD:    local[j] = fmod(a, b);
                            while (local[j] < 0)
                            {
                                local[j] += b;
                            }
                            break;
            case OP_NANFILL:    local[j] = isnan(a) ? b : a; break;

            case OP_SQUARE: local[j] = a * a; break;
            case OP_SQRT:   local[j] = sqrt(a); break;
            case OP_NEG:    local[j] = -a; break;
            case OP_ABS:    local[j] = fabs(a); break;
            case OP_SIN:    local[j] = sin(a); break;
            case OP_COS:    local[j] = cos(a); break;
            case OP_TAN:    local[j] = tan(a); break;
            case OP_ASIN:   local[j] = asin(a); break;
            case OP_ACOS:   local[j] = acos(a); break;
            case OP_ATAN:   local[j] = atan(a); break;
            case OP_EXP:    local[j] = exp(a); break;
        }
        j++;
    }

    // Collect the resulting value and put it into the output array
    out[index] = local[root];
}

////////////////////////////////////////////////////////////////////////////////

TapeAccelerator::TapeAccelerator(Evaluator* e)
    : Accelerator(e)
{
    // Allocate space for the input and output arrays
    hipMalloc((void**)&X_d, N * sizeof(float));
    hipMalloc((void**)&Y_d, N * sizeof(float));
    hipMalloc((void**)&Z_d, N * sizeof(float));
    hipMalloc((void**)&out_d, N * sizeof(float));


    // Construct the tape!
    std::vector<uint32_t> tape;

    tape.push_back(OP_X);
    tape.push_back(OP_Y);
    tape.push_back(OP_Z);
    clauses = 3;

    std::unordered_map<Clause*, uint32_t> addr =
        {{e->X, 0}, {e->Y, 1}, {e->Z, 2}};

    for (const auto& r : e->rows)
    {
        for (size_t i=0; i < r.active; ++i)
        {
            Clause* c = r[i];
            uint32_t op = c->op;
            addr[c] = clauses++;

            if (c->a)
            {
                if (c->a->op == OP_CONST)
                {
                    op |= ARG_A_IMM;
                }
                else
                {
                    op |= ARG_A_MEM;
                }
            }

            if (c->b)
            {
                if (c->b->op == OP_CONST)
                {
                    op |= ARG_B_IMM;
                }
                else
                {
                    op |= ARG_B_MEM;
                }
            }

            tape.push_back(op);
            if (op & ARG_A_IMM)
            {
                tape.push_back(*(uint32_t*)&c->a->value);
            }
            else if (op & ARG_A_MEM)
            {
                tape.push_back(addr[c->a]);
            }

            if (op & ARG_B_IMM)
            {
                tape.push_back(*(uint32_t*)&c->b->value);
            }
            else if (op & ARG_B_MEM)
            {
                tape.push_back(addr[c->b]);
            }
        }
    }
    tape.push_back(0);

    assert(addr.count(e->root));
    root = addr[e->root];

    // Allocate the tape and copy it over to the GPU
    size_t tape_bytes = tape.size() * sizeof(uint32_t);
    hipMalloc((void**)&tape_d, tape_bytes);
    hipMemcpy(tape_d, &tape[0], tape_bytes, hipMemcpyHostToDevice);

    // Allocate the working memory buffer
    // (it's empty to begin with and populated by the kernel)
    hipMalloc((void**)&mem_d, clauses * N * sizeof(float));
}

TapeAccelerator::~TapeAccelerator()
{
    for (auto& ptr : {X_d, Y_d, Z_d, out_d, out_d})
    {
        hipFree(ptr);
    }
    hipFree(tape_d);
}

float* TapeAccelerator::values(size_t count)
{
    int threads = 256;
    int blocks = (count + threads - 1) / threads;

    eval<<<blocks, threads>>>(tape_d, X_d, Y_d, Z_d, out_d, mem_d,
                              clauses, root);

    return out_d;
}
