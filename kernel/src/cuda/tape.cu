#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2016 Matthew Keeter  <matt.j.keeter@gmail.com>
 *
 *  This file is part of the Ao library.
 *
 *  This library is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU Lesser General Public
 *  License as published by the Free Software Foundation; either
 *  version 2.1 of the License, or (at your option) any later version.
 *
 *  Ao is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with Ao.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "ao/kernel/cuda/tape.hpp"

#include "ao/kernel/eval/evaluator.hpp"
#include "ao/kernel/eval/clause.hpp"

#define ARG_A_IMM 0x100
#define ARG_A_MEM 0x200

#define ARG_B_IMM 0x1000
#define ARG_B_MEM 0x2000
/*
 *  eval is a kernel that executes an instruction tape in parallel.
 *  The arguments are as follows:
 *      tape is a list of opcodes enums or'd with supporting data
 *          The lowest byte of each uint32 is the opcode itself
 *          The third and fourth nibbles are both
 *                  0x0 if this argument isn't relevant
 *                  0x1 if an immediate argument follows
 *                  0x2 if a memory address is given
 *              for the a and b arguments
 *      X, Y, Z are pointers into device memory storing coordinates
 *      out is a pointer into device memory for the output
 *      clauses is the number of clauses to be evaluated
 *      root is the clause number to be copied to output
 */
__global__ void eval(uint32_t const* tape_,
                     float const* X, float const* Y, float const* Z,
                     float* out, uint32_t tape_size, uint32_t root)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // This is our local slice of memory used to store clause results
    float local[TapeAccelerator::NUM_CLAUSES];

    // Use the power of friendship to quickly copy the tape into local memory
    __shared__ uint32_t tape[TapeAccelerator::NUM_CLAUSES * 3];
    {
        // How many items from the tape must each thread copy over?
        // (rounded up, so we'll need to truncate later)
        unsigned num = (tape_size + TapeAccelerator::THREADS_PER_BLOCK - 1) /
                        TapeAccelerator::THREADS_PER_BLOCK;

        // This is the starting point in tape memory assigned to this thread
        unsigned start = threadIdx.x * num;
        unsigned end = start + num;
        if (end > tape_size)
        {
            end = tape_size;
        }
        for (unsigned i=start; i < end; ++i)
        {
            tape[i] = tape_[i];
        }
    }
    __syncthreads();

    // Load coordinates into the buffer
    local[0] = X[index];
    local[1] = Y[index];
    local[2] = Z[index];

    // First three opcodes are dummies for X, Y, Z coordinates
    for (int tape_index=3, clause_index=3; tape_index < tape_size;
         tape_index++, clause_index++)
    {
        // Grab the next opcode from the tape
        uint32_t opcode = tape[tape_index];

        // These are the values that we'll do math on
        float a, b;

        // Read arguments if present, selecting either from an immediate
        // argument (i.e. an inline float) or an address in the local mem
        if (opcode & ARG_A_IMM)
        {
            a = ((float*)tape)[++tape_index];
        }
        else if (opcode & ARG_A_MEM)
        {
            a = local[tape[++tape_index]];
        }

        if (opcode & ARG_B_IMM)
        {
            b = ((float*)tape)[++tape_index];
        }
        else if (opcode & ARG_B_MEM)
        {
            b = local[tape[++tape_index]];
        }

        switch (opcode & 0xFF)
        {
            case OP_ADD:    local[clause_index] = a + b; break;
            case OP_MUL:    local[clause_index] = a * b; break;

            case OP_MIN:    local[clause_index] = fmin(a, b); break;
            case OP_MAX:    local[clause_index] = fmax(a, b); break;
            case OP_SUB:    local[clause_index] = a - b; break;
            case OP_DIV:    local[clause_index] = a / b; break;
            case OP_ATAN2:  local[clause_index] = atan2(a, b); break;
            case OP_MOD:    local[clause_index] = fmod(a, b);
                            while (local[clause_index] < 0)
                            {
                                local[clause_index] += b;
                            }
                            break;
            case OP_NANFILL:    local[clause_index] = isnan(a) ? b : a; break;

            case OP_SQUARE: local[clause_index] = a * a; break;
            case OP_SQRT:   local[clause_index] = sqrt(a); break;
            case OP_NEG:    local[clause_index] = -a; break;
            case OP_ABS:    local[clause_index] = fabs(a); break;
            case OP_SIN:    local[clause_index] = sin(a); break;
            case OP_COS:    local[clause_index] = cos(a); break;
            case OP_TAN:    local[clause_index] = tan(a); break;
            case OP_ASIN:   local[clause_index] = asin(a); break;
            case OP_ACOS:   local[clause_index] = acos(a); break;
            case OP_ATAN:   local[clause_index] = atan(a); break;
            case OP_EXP:    local[clause_index] = exp(a); break;
        }
    }

    // Collect the resulting value and put it into the output array
    out[index] = local[root];
}

////////////////////////////////////////////////////////////////////////////////

TapeAccelerator::TapeAccelerator(Evaluator* e)
    : Accelerator(e)
{
    // Allocate space for the input and output arrays
    hipMalloc((void**)&X_d, N * sizeof(float));
    hipMalloc((void**)&Y_d, N * sizeof(float));
    hipMalloc((void**)&Z_d, N * sizeof(float));
    hipMalloc((void**)&out_d, N * sizeof(float));

    // In the worst case, every clause in the tape has two operands
    hipMalloc((void**)&tape_d, NUM_CLAUSES * sizeof(uint32_t) * 3);

    reloadTape();
}

void TapeAccelerator::reloadTape()
{
    // Construct the tape!
    std::vector<uint32_t> tape;

    tape.push_back(OP_X);
    tape.push_back(OP_Y);
    tape.push_back(OP_Z);
    size_t clauses = 3;

    std::unordered_map<Clause*, uint32_t> addr =
        {{evaluator->X, 0}, {evaluator->Y, 1}, {evaluator->Z, 2}};

    for (const auto& r : evaluator->rows)
    {
        for (size_t i=0; i < r.active; ++i)
        {
            Clause* c = r[i];
            uint32_t op = c->op;
            addr[c] = clauses++;

            if (c->a)
            {
                if (c->a->op == OP_CONST)
                {
                    op |= ARG_A_IMM;
                }
                else
                {
                    op |= ARG_A_MEM;
                }
            }

            if (c->b)
            {
                if (c->b->op == OP_CONST)
                {
                    op |= ARG_B_IMM;
                }
                else
                {
                    op |= ARG_B_MEM;
                }
            }

            tape.push_back(op);
            if (op & ARG_A_IMM)
            {
                tape.push_back(*(uint32_t*)&c->a->value);
            }
            else if (op & ARG_A_MEM)
            {
                tape.push_back(addr[c->a]);
            }

            if (op & ARG_B_IMM)
            {
                tape.push_back(*(uint32_t*)&c->b->value);
            }
            else if (op & ARG_B_MEM)
            {
                tape.push_back(addr[c->b]);
            }
        }
    }

    assert(addr.count(evaluator->root));
    root = addr[evaluator->root];

    // Save the tape's size and copy it over to the GPU
    tape_size = tape.size();
    size_t tape_bytes = tape_size * sizeof(uint32_t);
    hipMemcpy(tape_d, &tape[0], tape_bytes, hipMemcpyHostToDevice);
}

TapeAccelerator::~TapeAccelerator()
{
    for (auto& ptr : {X_d, Y_d, Z_d, out_d, out_d})
    {
        hipFree(ptr);
    }
    hipFree(tape_d);
}

float* TapeAccelerator::values(size_t count)
{
    int blocks = (count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    eval<<<blocks, THREADS_PER_BLOCK>>>(
            tape_d, X_d, Y_d, Z_d, out_d, tape_size, root);

    return out_d;
}
