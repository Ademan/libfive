#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2016 Matthew Keeter  <matt.j.keeter@gmail.com>
 *
 *  This file is part of the Ao library.
 *
 *  This library is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU Lesser General Public
 *  License as published by the Free Software Foundation; either
 *  version 2.1 of the License, or (at your option) any later version.
 *
 *  Ao is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with Ao.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "ao/kernel/cuda/accelerator.hpp"
#include "ao/kernel/render/subregion.hpp"

__global__ void flatten_region(float* x, float xmin, float xmax, int ni,
                               float* y, float ymin, float ymax, int nj,
                               float* z, float zmin, float zmax, int nk)
{
    int index = blockIdx.x;
    int i = index / (nj * nk);
    int j = (index / nk) % nj;
    int k = index % nk;

    if (i <= ni)
    {
        float frac = (i + 0.5f) / ni;
        x[index] = xmin * (1.0f - frac) + xmax * frac;
    }
    if (j <= nj)
    {
        float frac = (j + 0.5f) / nj;
        y[index] = ymin * (1.0f - frac) + ymax * frac;
    }
    if (k <= nk)
    {
        float frac = (k + 0.5f) / nk;
        z[index] = zmin * (1.0f - frac) + zmax * frac;
    }
}

////////////////////////////////////////////////////////////////////////////////

Accelerator::Accelerator(Evaluator* e)
    : evaluator(e)
{

}

float* Accelerator::fromDevice(float* ptr_d)
{
    const size_t bytes = N * sizeof(float);
    hipMemcpy(&buf[0], ptr_d, bytes, hipMemcpyDeviceToHost);
    return &buf[0];
}

__global__ void _warmup(float* a, float * b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    b[i] = a[i] * 2;
}

void Accelerator::warmup()
{
    float* a = new float[N];
    float* b = new float[N];

    float* a_d = nullptr;
    float* b_d = nullptr;

    auto bytes = N * sizeof(float);
    hipMalloc((void**)&a_d, bytes);
    hipMalloc((void**)&b_d, bytes);

    hipMemcpy(&a_d, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(&b_d, b, bytes, hipMemcpyHostToDevice);

    _warmup<<<128, 32>>>(a, b);
    hipMemcpy(b, b_d, bytes, hipMemcpyDeviceToHost);

    delete [] a;
    delete [] b;

    hipFree(a_d);
    hipFree(b_d);
}

void Accelerator::toDevice()
{
    const size_t bytes = N * sizeof(float);
    hipMemcpy(X_d, &X[0], bytes, hipMemcpyHostToDevice);
    hipMemcpy(Y_d, &Y[0], bytes, hipMemcpyHostToDevice);
    hipMemcpy(Z_d, &Z[0], bytes, hipMemcpyHostToDevice);
}


void Accelerator::setRegion(const Subregion& r)
{
    flatten_region<<<r.voxels(), 1>>>(
        X_d, r.X.lower(), r.X.upper(), r.X.size,
        Y_d, r.Y.lower(), r.Y.upper(), r.Y.size,
        Z_d, r.Z.lower(), r.Z.upper(), r.Z.size);
}
