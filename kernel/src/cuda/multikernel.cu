#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2016 Matthew Keeter  <matt.j.keeter@gmail.com>
 *
 *  This file is part of the Ao library.
 *
 *  This library is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU Lesser General Public
 *  License as published by the Free Software Foundation; either
 *  version 2.1 of the License, or (at your option) any later version.
 *
 *  Ao is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with Ao.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <cstdio>
#include <numeric>

#include "ao/kernel/cuda/multikernel.hpp"

#include "ao/kernel/eval/evaluator.hpp"
#include "ao/kernel/eval/clause.hpp"

// Helpful macros that are repeated in every kernel
#define GET_INDEX int i = threadIdx.x + blockIdx.x * blockDim.x;\
                  if (i >= MultikernelAccelerator::N)  return
#define KERNEL(name) __global__ void name(float* __restrict__ a, float* __restrict__ b, float* __restrict__ out)

////////////////////////////////////////////////////////////////////////////////
// Floating-point kernels
KERNEL(add_f)
{
    GET_INDEX;
    out[i] = a[i] + b[i];
}

KERNEL(mul_f)
{
    GET_INDEX;
    out[i] = a[i] * b[i];
}

KERNEL(min_f)
{
    GET_INDEX;
    out[i] = min(a[i], b[i]);
}

KERNEL(max_f)
{
    GET_INDEX;
    out[i] = max(a[i], b[i]);
}

KERNEL(sub_f)
{
    GET_INDEX;
    out[i] = a[i] - b[i];
}

KERNEL(div_f)
{
    GET_INDEX;
    out[i] = a[i] / b[i];
}

KERNEL(atan2_f)
{
    GET_INDEX;
    out[i] = atan2(a[i], b[i]);
}

KERNEL(mod_f)
{
    GET_INDEX;
    out[i] = fmod(a[i], b[i]);
    while (out[i] < 0)
    {
        out[i] += b[i];
    }
}

KERNEL(nanfill_f)
{
    GET_INDEX;
    out[i] = isnan(a[i]) ? b[i] : a[i];
}

KERNEL(square_f)
{
    GET_INDEX;
    out[i] = a[i] * a[i];
}

KERNEL(sqrt_f)
{
    GET_INDEX;
    out[i] = sqrt(a[i]);
}

KERNEL(neg_f)
{
    GET_INDEX;
    out[i] = -a[i];
}

KERNEL(abs_f)
{
    GET_INDEX;
    out[i] = fabs(a[i]);
}

KERNEL(sin_f)
{
    GET_INDEX;
    out[i] = sin(a[i]);
}

KERNEL(cos_f)
{
    GET_INDEX;
    out[i] = cos(a[i]);
}

KERNEL(tan_f)
{
    GET_INDEX;
    out[i] = tan(a[i]);
}

KERNEL(asin_f)
{
    GET_INDEX;
    out[i] = asin(a[i]);
}

KERNEL(acos_f)
{
    GET_INDEX;
    out[i] = acos(a[i]);
}

KERNEL(atan_f)
{
    GET_INDEX;
    out[i] = atan(a[i]);
}

KERNEL(exp_f)
{
    GET_INDEX;
    out[i] = exp(a[i]);
}

KERNEL(a_f)
{
    GET_INDEX;
    out[i] = a[i];
}

KERNEL(b_f)
{
    GET_INDEX;
    out[i] = b[i];
}

////////////////////////////////////////////////////////////////////////////////

// Pointers are into device memory, not host memory!
static void clause(Opcode op, float* a, float* b, float* out,
                   int blocks, int threads)
{
    switch (op) {
        case OP_ADD:    add_f<<<blocks, threads>>>(a, b, out); break;
        case OP_MUL:    mul_f<<<blocks, threads>>>(a, b, out); break;
        case OP_MIN:    min_f<<<blocks, threads>>>(a, b, out); break;
        case OP_MAX:    max_f<<<blocks, threads>>>(a, b, out); break;
        case OP_SUB:    sub_f<<<blocks, threads>>>(a, b, out); break;
        case OP_DIV:    div_f<<<blocks, threads>>>(a, b, out); break;
        case OP_ATAN2:  atan2_f<<<blocks, threads>>>(a, b, out); break;
        case OP_MOD:    mod_f<<<blocks, threads>>>(a, b, out); break;
        case OP_NANFILL:    nanfill_f<<<blocks, threads>>>(a, b, out); break;

        case OP_SQUARE: square_f<<<blocks, threads>>>(a, b, out); break;
        case OP_SQRT: sqrt_f<<<blocks, threads>>>(a, b, out); break;
        case OP_NEG: neg_f<<<blocks, threads>>>(a, b, out); break;
        case OP_ABS: abs_f<<<blocks, threads>>>(a, b, out); break;
        case OP_SIN: sin_f<<<blocks, threads>>>(a, b, out); break;
        case OP_COS: cos_f<<<blocks, threads>>>(a, b, out); break;
        case OP_TAN: tan_f<<<blocks, threads>>>(a, b, out); break;
        case OP_ASIN: asin_f<<<blocks, threads>>>(a, b, out); break;
        case OP_ACOS: acos_f<<<blocks, threads>>>(a, b, out); break;
        case OP_ATAN: atan_f<<<blocks, threads>>>(a, b, out); break;
        case OP_EXP: exp_f<<<blocks, threads>>>(a, b, out); break;

        case OP_A: a_f<<<blocks, threads>>>(a, b, out); break;
        case OP_B: b_f<<<blocks, threads>>>(a, b, out); break;

        case INVALID:
        case OP_CONST:
        case OP_X:
        case OP_Y:
        case OP_Z:
        case AFFINE:
        case LAST_OP: assert(false);
    }
}

////////////////////////////////////////////////////////////////////////////////

MultikernelAccelerator::MultikernelAccelerator(Evaluator* e)
    : Accelerator(e)
{
    // Count up the number of clauses in the evaluator
    size_t count =  std::accumulate(e->rows.begin(), e->rows.end(),
            3                           // X, Y, Z
            + e->constants.size(),      // Constants
            [](size_t i, const Row& r){ return i + r.size(); });

    auto out = hipMalloc((void**)&data, N * count * sizeof(float));

    // Populate X, Y, Z device pointers
    X_d = devPtr(evaluator->X);
    Y_d = devPtr(evaluator->Y);
    Z_d = devPtr(evaluator->Z);

    // If this is a constant operation, fill with the constant value
    for (auto c : e->constants)
    {
        std::fill(buf.begin(), buf.end(), c->value);
        hipMemcpy(devPtr(c), &buf[0], N * sizeof(float),
                   hipMemcpyHostToDevice);
    }
}

MultikernelAccelerator::~MultikernelAccelerator()
{
    hipFree(data);
}

////////////////////////////////////////////////////////////////////////////////

float* MultikernelAccelerator::devPtr(const Clause* c)
{
    // Clauses are allocated in the data array of the evaluator
    // (using placement new); this function finds the clause's
    // location in that array then maps to the device data array
    return &data[(c - evaluator->data) * N];
}

////////////////////////////////////////////////////////////////////////////////

float* MultikernelAccelerator::values(size_t count)
{
    int threads = 256;
    int blocks = (count + threads - 1) / threads;

    for (const auto& row : evaluator->rows)
    {
        for (size_t i=0; i < row.active; ++i)
        {
            auto op = row[i]->op;

            // Modify the opcode if parts of the tree are disabled
            if (row[i]->a && row[i]->a->flags & CLAUSE_FLAG_DISABLED)
            {
                op = OP_B;
            }
            if (row[i]->b && row[i]->b->flags & CLAUSE_FLAG_DISABLED)
            {
                op = OP_A;
            }
            clause(op, devPtr(row[i]->a), devPtr(row[i]->b), devPtr(row[i]),
                   blocks, threads);
        }
    }

    return devPtr(evaluator->root);
}
