#include "hip/hip_runtime.h"
#include <cstdio>

#include "ao/kernel/render/accelerator.hpp"
#include "ao/kernel/eval/evaluator.hpp"
#include "ao/kernel/eval/clause.hpp"

__global__ void mykernel(void)
{
    // Nothing to do here
}

int run(void)
{
    mykernel<<<1,1>>>();
    printf("Hello World!\n");
    return 0;
}

Accelerator::Accelerator(Evaluator* e)
{
    for (auto c : {e->X, e->Y, e->Z})
    {
        allocate(c);
    }
    for (auto c : e->matrix)
    {
        allocate(c);
    }
    for (auto c : e->constants)
    {
        allocate(c);
    }
    for (auto row : e->rows)
    {
        for (auto c : row)
        {
            allocate(c);
        }
    }
}

void Accelerator::allocate(Clause* c)
{
    hipMalloc((void**)&mem[c], N * sizeof(float));
}
